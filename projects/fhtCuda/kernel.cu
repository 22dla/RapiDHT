#include "hip/hip_runtime.h"
﻿#include "kernel.h"
#include ""
#include "dev_array.h"

//using DataType = unsigned __int8;

__global__ void matrixMultiplicationKernel(DataType* A, DataType* B, DataType* C, int N) {

	int ROW = blockIdx.y*blockDim.y + threadIdx.y;
	int COL = blockIdx.x*blockDim.x + threadIdx.x;

	DataType tmpSum = 0;

	if (ROW < N && COL < N) {
		// each thread computes one element of the block sub-matrix
		for (int i = 0; i < N; i++) {
			tmpSum += A[ROW * N + i] * B[i * N + COL];
		}
	}
	C[ROW * N + COL] = tmpSum;
}

void matrixMultiplication(DataType *A, DataType *B, DataType *C, int N) {

	// declare the number of blocks per grid and the number of threads per block
	// use 1 to 512 threads per block
	dim3 threadsPerBlock(N, N);
	dim3 blocksPerGrid(1, 1);
	if (N*N > 512) {
		threadsPerBlock.x = 512;
		threadsPerBlock.y = 512;
		blocksPerGrid.x = ceil(double(N) / double(threadsPerBlock.x));
		blocksPerGrid.y = ceil(double(N) / double(threadsPerBlock.y));
	}

	matrixMultiplicationKernel <<< blocksPerGrid, threadsPerBlock >>> (A, B, C, N);
}
