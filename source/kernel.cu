#include "hip/hip_runtime.h"
﻿#include "kernel.h"
#include ""
#include "dev_array.h"

// ------------------------------ Kernels ------------------------------

__global__ void matrixMultiplicationKernel(const double* A, const double* B, double* C, int M, int K, int N) {
	int row = blockIdx.y * blockDim.y + threadIdx.y; // индекс строки C
	int col = blockIdx.x * blockDim.x + threadIdx.x; // индекс столбца C

	if (row < M && col < N) {
		double sum = 0.0;
		for (int t = 0; t < K; ++t) {
			sum += A[row * K + t] * B[t * N + col];
		}
		C[row * N + col] = sum;
	}
}

__global__ void matrixVectorMultKernel(const double* A, const double* x, double* y, int N) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < N) {
		double sum = 0.0;
		for (int j = 0; j < N; j++) {
			sum += A[i * N + j] * x[j];
		}
		y[i] = sum;
	}
}

__global__ void matrixTransposeKernel(double* A, int N) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;

	if (i < N && j < N && i < j) {
		double tmp = A[i * N + j];
		A[i * N + j] = A[j * N + i];
		A[j * N + i] = tmp;
	}
}

__global__ void matrixTransposeKernel(const double* A, double* B, int rows, int cols) {
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	int row = blockIdx.y * blockDim.y + threadIdx.y;

	if (col < cols && row < rows) {
		B[col * rows + row] = A[row * cols + col];
	}
}

// ------------------------------ Host Wrappers ------------------------------

void matrixMultiplication(const double* A, const double* B, double* C, int M, int K, int N) {
	const int BLOCK_SIZE = 16;
	dim3 threadsPerBlock(BLOCK_SIZE, BLOCK_SIZE);
	dim3 blocksPerGrid(
		(N + BLOCK_SIZE - 1) / BLOCK_SIZE,
		(M + BLOCK_SIZE - 1) / BLOCK_SIZE);

	matrixMultiplicationKernel << <blocksPerGrid, threadsPerBlock >> > (A, B, C, M, K, N);

	hipDeviceSynchronize();
}

void matrixMultiplication(const double* A, const double* B, double* C, int N) {
	matrixMultiplication(A, B, C, N, N, N);
}

void vectorMatrixMultiplication(const double* A, const double* x, double* y, int N) {
	int threadsPerBlock = (N > 512) ? 512 : N;
	int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;

	matrixVectorMultKernel << < blocksPerGrid, threadsPerBlock >> > (A, x, y, N);
	hipDeviceSynchronize();
}

// rows, cols - целевые (размеры матрицы B)
void matrixTranspose(const double* A, double* B, int rows, int cols) {
	int BLOCK_SIZE = 16;
	dim3 threadsPerBlock(BLOCK_SIZE, BLOCK_SIZE);
	dim3 blocksPerGrid((cols + BLOCK_SIZE - 1) / BLOCK_SIZE,
		(rows + BLOCK_SIZE - 1) / BLOCK_SIZE);

	matrixTransposeKernel << <blocksPerGrid, threadsPerBlock >> > (A, B, rows, cols);
	hipDeviceSynchronize();
}

void matrixTranspose(double* A, int N) {
	int BLOCK_SIZE = 16;  // оптимальный размер блока
	dim3 threadsPerBlock(BLOCK_SIZE, BLOCK_SIZE);
	dim3 blocksPerGrid((N + BLOCK_SIZE - 1) / BLOCK_SIZE,
		(N + BLOCK_SIZE - 1) / BLOCK_SIZE);

	matrixTransposeKernel << < blocksPerGrid, threadsPerBlock >> > (A, N);
	hipDeviceSynchronize();
}
