#include "hip/hip_runtime.h"
﻿#include "kernel.h"
#include ""
#include "dev_array.h"

//using DataType = unsigned __int8;

__global__ void matrixMultiplicationKernel(float* A, float* B, float* C, int N) {

	int ROW = blockIdx.y*blockDim.y + threadIdx.y;
	int COL = blockIdx.x*blockDim.x + threadIdx.x;

	float tmpSum = 0;

	if (ROW < N && COL < N) {
		// each thread computes one element of the block sub-matrix
		for (int i = 0; i < N; i++) {
			//tmpSum += A[ROW * N + i] * B[i];			// for A * b = c (b, c - vectors)
			tmpSum += A[ROW * N + i] * B[i * N + COL];	// for A * B = C (b, c - matrices)
		}
	}
	//C[ROW] = tmpSum;									// for A * b = c (b, c - vectors)
	C[ROW * N + COL] = tmpSum;							// for A * B = C (b, c - matrices)
}

__global__ void matrixVectorMult(float* A, float* x, float* y, int N) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < N) {
		float sum = 0.0f;
		for (int j = 0; j < N; j++) {
			sum += A[i * N + j] * x[j];
		}
		y[i] = sum;
	}
}

void matrixMultiplication(float *A, float *B, float *C, int N) {

	// declare the number of blocks per grid and the number of threads per block
	// use 1 to 512 threads per block
	dim3 threadsPerBlock(N, N);
	dim3 blocksPerGrid(1, 1);
	if (N*N > 512) {
		threadsPerBlock.x = 512;
		threadsPerBlock.y = 512;
		blocksPerGrid.x = ceil(double(N) / double(threadsPerBlock.x));
		blocksPerGrid.y = ceil(double(N) / double(threadsPerBlock.y));
	}

	matrixMultiplicationKernel <<< blocksPerGrid, threadsPerBlock >>> (A, B, C, N);
}

void vectorMatrixMultiplication(float* A, float* x, float* y, int N) {

	int threadsPerBlock, blocksPerGrid;

	threadsPerBlock = (N > 512) ? 512 : N;
	blocksPerGrid = (N > 512) ? 1 : (N + threadsPerBlock - 1) / threadsPerBlock;

	matrixVectorMult <<<blocksPerGrid, threadsPerBlock >>> (A, x, y, N);
}
